
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void kernel() {
    printf("Hello from the GPU!\n");
}

int main() {
    kernel<<<1,1>>>();
    hipDeviceSynchronize();
    return 0;
}